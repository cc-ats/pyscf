/*
  simple wrapper to utility cuda routines  
 */

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>


extern "C" int CountDevices()
{
  int num_gpus = -1;
  hipGetDeviceCount(&num_gpus);
  return num_gpus;
}

extern "C" void SetDevice(int gpu_id)
{
  hipSetDevice(gpu_id);
}

extern "C" int GetDevice()
{
  int gpu_id = -1;
  hipGetDevice(&gpu_id);
  return gpu_id;
}
